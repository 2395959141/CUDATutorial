#include <stdio.h>
#include <hip/hip_runtime.h>


typedef float FLOAT;

/* CUDA kernel function */
__global__ void vec_add(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    /* 2D grid */
    int idx = (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x);
    /* 1D grid */
    // int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < N) z[idx] = y[idx] + x[idx];
}

void vec_add_cpu(FLOAT *x, FLOAT *y, FLOAT *z, int N)
{
    for (int i = 0; i < N; i++) z[i] = y[i] + x[i];
}

int main()
{
    int N = 10000;
    int nbytes = N * sizeof(FLOAT);

    /* 1D block */
    int bs = 256;

    /* 2D grid */
    int s = ceil(sqrt((N + bs - 1.) / bs));  //用来向上取整的技巧，计算网络中需要多少块
    dim3 grid(s, s);   //使用 dim3 类型定义了一个二维网格，其大小为 s x s，
                       //这意味着这个网格有 s 行和 s 列，每个位置上都是一个 block。
    
    /* 1D grid */
    // int s = ceil((N + bs - 1.) / bs);
    // dim3 grid(s);

    // FLOAT *dx, *hx;
    // FLOAT *dy, *hy;
    // FLOAT *dz, *hz;

    FLOAT *hx, *hy, *hz;

    // /* allocate GPU mem */
    // cudaMalloc((void **)&dx, nbytes);
    // cudaMalloc((void **)&dy, nbytes);
    // cudaMalloc((void **)&dz, nbytes);
    
    // /* init time */
    // float milliseconds = 0;

    // /* alllocate CPU mem */
    // hx = (FLOAT *) malloc(nbytes);
    // hy = (FLOAT *) malloc(nbytes);
    // hz = (FLOAT *) malloc(nbytes);

    // /* init */
    // for (int i = 0; i < N; i++) {
    //     hx[i] = 1;
    //     hy[i] = 1;
    // }

    // /* copy data to GPU */
    // cudaMemcpy(dx, hx, nbytes, cudaMemcpyHostToDevice);
    // cudaMemcpy(dy, hy, nbytes, cudaMemcpyHostToDevice);

    /* 使用 cudaMallocManaged 进行统一内存分配 */
    hipMallocManaged(&hx, nbytes);
    hipMallocManaged(&hy, nbytes);
    hipMallocManaged(&hz, nbytes);

     /* 初始化数据 */
    for (int i = 0; i < N; i++) {
        hx[i] = 1;
        hy[i] = 1;
    }

     /* init time */
    float milliseconds = 0;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start);
    /* launch GPU kernel */
    vec_add<<<grid, bs>>>(hx, hy, hz, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);  
    
	// /* copy GPU result to CPU */
    // cudaMemcpy(hz, dz, nbytes, cudaMemcpyDeviceToHost);

    /* CPU compute */
    FLOAT* hz_cpu_res = (FLOAT *) malloc(nbytes);
    vec_add_cpu(hx, hy, hz_cpu_res, N);

    /* check GPU result with CPU*/
    for (int i = 0; i < N; ++i) {
        if (fabs(hz_cpu_res[i] - hz[i]) > 1e-6) {
            printf("Result verification failed at element index %d!\n", i);
        }
    }
    printf("Result right\n");
    printf("Mem BW= %f (GB/sec)\n", (float)N*4/milliseconds/1e6); ///
    // cudaFree(dx);
    // cudaFree(dy);
    // cudaFree(dz);

    hipFree(hx);
    hipFree(hy);
    hipFree(hz);

    free(hz_cpu_res);

    return 0;
}