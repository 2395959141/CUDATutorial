#include <stdio.h>
#include <hip/hip_runtime.h>


#define ARRAY_SIZE 100000000   //Array size has to exceed L2 size to avoid L2 cache residence
#define MEMORY_OFFSET 10000000
#define BENCH_ITER 10
#define THREADS_NUM 256
//global memory bandwidth = 349Gb/s
//float4 vectoradd
__global__ void mem_bw (float* A,  float* B, float* C){
	// block and thread index
	//int idx = blockIdx.x * blockDim.x + threadIdx.x;
	int idx = blockIdx.x * blockDim.x * 4 + threadIdx.x;
	for(int i = idx; i < MEMORY_OFFSET / 4; i += blockDim.x * gridDim.x) {
		float4 a1 = reinterpret_cast<float4*>(A)[i];
		float4 b1 = reinterpret_cast<float4*>(B)[i];
		float4 c1;

		c1.x = a1.x + b1.x;
		c1.y = a1.y + b1.y;
		c1.z = a1.z + b1.z;
		c1.w = a1.w + b1.w;
		reinterpret_cast<float4*>(C)[i] = c1;
	}
}

void vec_add_cpu(float *x, float *y, float *z, int N)
{
    for (int i = 0; i < 20; i++) z[i] = y[i] + x[i];
}

int main(){
	float *A = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *B = (float*) malloc(ARRAY_SIZE*sizeof(float));
	float *C = (float*) malloc(ARRAY_SIZE*sizeof(float));

	float *A_g;
	float *B_g;
	float *C_g;

	float milliseconds = 0;

	for (uint32_t i=0; i<ARRAY_SIZE; i++){
		A[i] = (float)i;
		B[i] = (float)i;
	}
	hipMalloc((void**)&A_g, ARRAY_SIZE*sizeof(float));
	hipMalloc((void**)&B_g, ARRAY_SIZE*sizeof(float));
	hipMalloc((void**)&C_g, ARRAY_SIZE*sizeof(float));

	hipMemcpy(A_g, A, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_g, B, ARRAY_SIZE*sizeof(float), hipMemcpyHostToDevice);
  
	int BlockNums = MEMORY_OFFSET / 256;
    //warm up to occupy L2 cache
	printf("warm up start\n");
	mem_bw<<<BlockNums, THREADS_NUM>>>(A_g, B_g, C_g);
	printf("warm up end\n");
    // time start using cudaEvent
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start);
	for (int i = BENCH_ITER - 1; i >= 0; --i) {
		mem_bw<<<BlockNums, THREADS_NUM>>>(A_g + i * MEMORY_OFFSET, B_g + i * MEMORY_OFFSET, C_g + i * MEMORY_OFFSET);
	}
	// time stop using cudaEvent
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&milliseconds, start, stop);

	hipMemcpy(C, C_g, ARRAY_SIZE*sizeof(float), hipMemcpyDeviceToHost);
	/* CPU compute */
	float* C_cpu_res = (float *) malloc(20*sizeof(float));
	vec_add_cpu(A, B, C_cpu_res, ARRAY_SIZE);

	/* check GPU result with CPU*/
	for (int i = 0; i < 20; ++i) {
		if (fabs(C_cpu_res[i] - C[i]) > 1e-6) {
			printf("Result verification failed at element index %d!\n", i);
		}
	}
	printf("Result right\n");
	unsigned N = ARRAY_SIZE * 4;

	printf("Mem BW= %f (GB/sec)\n", 2 * (float)N / milliseconds / 1e6);//348g/sec达到T4 320+Gb/sec
}
