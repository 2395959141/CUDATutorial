#include <stdio.h>
#include <hip/hip_runtime.h>


template<typename T>
struct MaskScaleAndElementwiseAddFunctor {
  MaskScaleAndElementwiseAddFunctor(const uint8_t* mask, const T* add_val, float scale)
      : mask(mask), add_val(add_val), scale(scale) {}
  __device__ T Compute(T x, int64_t i) const {
    return x * static_cast<T>(static_cast<bool>(mask[i]) * scale) + add_val[i];
  }
  const uint8_t* mask;
  const T* add_val;
  float scale;
};

template<typename FUNCTOR, typename T>
__global__ void FusedBiasAddCUDAKernelFloat(FUNCTOR functor, const int elem_cnt, const int bias_size,
                                const T* x, const T* bias, T* y) {
  for (int i = blockIdx.x * blockDim.x + threadIdx.x; i < elem_cnt;
       i += blockDim.x * gridDim.x){
    T x_i = x[i] + bias[i % bias_size];
    y[i] = functor.Compute(x_i, i);
  }
}

int main(){
    constexpr int ele_cnt = 1000;
    float scale = 0.5;
    uint8_t* mask_tensor = new uint8_t[ele_cnt];
    float* add_val = new float[ele_cnt];
    for (int i = 0; i < ele_cnt; i++){
        mask_tensor[i] = (uint8_t)(i);
        add_val[i] = (float)(i);
    }
    int bias_size = 10;
    float *x = (float*) malloc(sizeof(float) * ele_cnt);
    float *y = (float*) malloc(sizeof(float) * ele_cnt);
    float *bias = (float*) malloc(sizeof(float) * bias_size);
    for (int i = 0; i < ele_cnt; i++)
    {
      x[i] = (float)(i);
    }
    float *d_x, *d_y, *d_bias;
    hipMalloc((void **)&d_x, ele_cnt * sizeof(float));
    hipMalloc((void **)&d_y, ele_cnt * sizeof(float));
    hipMalloc((void **)&d_bias, bias_size * sizeof(float));
    hipMemcpy(d_x, x, sizeof(float) * ele_cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_y, y, sizeof(float) * ele_cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_bias, bias, sizeof(float) * bias_size, hipMemcpyHostToDevice);
    uint8_t *d_mask_tensor;
    float *d_add_val;
    hipMalloc((void **)&d_mask_tensor, ele_cnt * sizeof(uint8_t));
    hipMalloc((void **)&d_add_val, ele_cnt * sizeof(float));
    hipMemcpy(d_mask_tensor, mask_tensor, sizeof(float) * ele_cnt, hipMemcpyHostToDevice);
    hipMemcpy(d_add_val, add_val, sizeof(float) * ele_cnt, hipMemcpyHostToDevice);

    hipDeviceProp_t deviceProp;
    hipGetDeviceProperties(&deviceProp, 0);
    int maxblocks = deviceProp.maxGridSize[0];
    int blockSize = 256;
    int gridSize = std::min((ele_cnt + blockSize - 1) / blockSize, maxblocks);
    MaskScaleAndElementwiseAddFunctor<float> mask_scale_and_elementwise_add_func(mask_tensor, add_val, scale);
    FusedBiasAddCUDAKernelFloat<<<gridSize , blockSize>>>(mask_scale_and_elementwise_add_func, ele_cnt, bias_size, d_x, d_bias, d_y);
    hipMemcpy(y, d_y, sizeof(float) * ele_cnt, hipMemcpyDeviceToHost);

    free(x);
    free(y);
    free(bias);
    delete add_val;
    add_val = nullptr;
    delete mask_tensor;
    add_val = nullptr;
    hipFree(d_x);
    hipFree(d_y);
    hipFree(d_bias);
    hipFree(d_mask_tensor);
    hipFree(d_add_val);
}
